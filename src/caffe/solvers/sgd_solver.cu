#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void SGDUpdate(int N, Dtype* g, Dtype* h,
    Dtype momentum, Dtype local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    g[i] = h[i] = momentum*h[i] + local_rate*g[i];
  }
}
template <typename Dtype>
void sgd_update_gpu(int N, Dtype* g, Dtype* h, Dtype momentum,
    Dtype local_rate) {
  SGDUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, g, h, momentum, local_rate);
  CUDA_POST_KERNEL_CHECK;
}
template void sgd_update_gpu<float>(int, float*, float*, float, float);
template void sgd_update_gpu<double>(int, double*, double*, double, double);

template <typename Dtype>
__global__ void abs_min_filter_kernel(int N, Dtype* a, Dtype min) {
  CUDA_KERNEL_LOOP(i, N) {
    if (abs(a[i]) < min) a[i] = 0;
  }
}
template <typename Dtype>
void abs_min_filter_gpu(int N, Dtype* a, Dtype min) {
  abs_min_filter_kernel<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, min);
  CUDA_POST_KERNEL_CHECK;
}
template void abs_min_filter_gpu<float>(int, float*, float);
template void abs_min_filter_gpu<double>(int, double*, double);

template <typename Dtype>
__global__ void set_mask_gpu_kernel(int N, const Dtype* a, Dtype min, Dtype* mask) {
  CUDA_KERNEL_LOOP(i, N) {
    if (abs(a[i]) < min)
      mask[i] = Dtype(0);
    else
      mask[i] = Dtype(1);
  }
}
template <typename Dtype>
void set_mask_gpu(int N, const Dtype* a, Dtype min, Dtype* mask) {
  set_mask_gpu_kernel<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, min, mask);
  CUDA_POST_KERNEL_CHECK;
}
template void set_mask_gpu<float>(int, const float*, float, float*);
template void set_mask_gpu<double>(int, const double*, double, double*);

}  // namespace caffe
